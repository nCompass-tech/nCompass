#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipblas.h>
#include <assert.h>
using namespace std;

#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
    fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
    exit(-1);}} while(0)
#define CUDA_CALL(X) ERR_NE((X),hipSuccess)
#define CUSPARSE_CALL(X) ERR_NE((X),HIPSPARSE_STATUS_SUCCESS)

template<class T>
struct reCuBuffer
{
    T* data = NULL;
    int len = 0;
};

template<class T>
void resize(reCuBuffer<T>& buffer, int size)
{
    if(size > buffer.len)
    {
        if(buffer.len > 0)
            CUDA_CALL(hipFree(buffer.data));
            
        CUDA_CALL(hipMalloc( &(buffer.data), size));
        buffer.len = size;
    }
}

#define num_device 16

static reCuBuffer<int>   nnzPerCol_[num_device], ColInd_[num_device], RowPtr_[num_device];
static reCuBuffer<float> csrVal_[num_device], tranBuffer_[num_device];
static reCuBuffer<void>  dBuffer_[num_device];

struct cublasHandle_
{
    hipblasHandle_t handle_;
    bool init = false;
};
static cublasHandle_ handle2_[num_device];

void sparse_mm_dense_cusparse_backend(const int & cuda_device_id, const int & m, const int & n, const int & p, float * dA, float * dB, float * dC)
{
    assert(cuda_device_id>=0);
    hipSetDevice(cuda_device_id);

    reCuBuffer<int>& nnzPerCol    = nnzPerCol_[cuda_device_id];
    reCuBuffer<int>& ColInd       = ColInd_[cuda_device_id];
    reCuBuffer<int>& RowPtr       = RowPtr_[cuda_device_id];
    reCuBuffer<float>& csrVal     = csrVal_[cuda_device_id];

    hipsparseHandle_t  handle;
    CUSPARSE_CALL(hipsparseCreate(&handle));

    // transform dense A to csr
    hipsparseMatDescr_t descrX;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrX));

    int total_nnz;
    resize(nnzPerCol, m * sizeof(int));

    CUSPARSE_CALL(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_COLUMN, n, m, descrX, dA, n, nnzPerCol.data, &total_nnz));
    resize(csrVal, total_nnz * sizeof(float));
    resize(ColInd, total_nnz * sizeof(int));
    resize(RowPtr, (m+1) * sizeof(int));

    CUSPARSE_CALL(hipsparseSdense2csc(handle, n, m, descrX, dA, n, nnzPerCol.data, csrVal.data, ColInd.data, RowPtr.data));

#if __CUDACC_VER_MAJOR__ == 10
    reCuBuffer<float>& tranBuffer = tranBuffer_[cuda_device_id];

    // CT = A * BT
    resize(tranBuffer, m * p * sizeof(float));

    // B * C
    hipsparseMatDescr_t descrA;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_CALL(hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO));

    float alpha = 1.0f;
    float beta  = 0.0f;
    CUSPARSE_CALL(hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_TRANSPOSE,
                  m,p,n,total_nnz,&alpha,descrA,csrVal.data,RowPtr.data, ColInd.data,dB,p,&beta,tranBuffer.data,m));
    CUSPARSE_CALL(hipsparseDestroyMatDescr(descrA));

    // hipblasDestroy will synchronize the device
    hipblasHandle_t& handle2 = handle2_[cuda_device_id].handle_;
    if(!handle2_[cuda_device_id].init)
    {
        hipblasCreate(&handle2);
        handle2_[cuda_device_id].init = true;
    }

    // C need TRANSPOSE
    hipblasSgeam(handle2, HIPBLAS_OP_T, HIPBLAS_OP_T, p, m, &alpha, tranBuffer.data, m, &beta, tranBuffer.data, m, dC, p);
    //hipblasDestroy(handle2);
#endif

#if __CUDACC_VER_MAJOR__ == 11
    reCuBuffer<void>& dBuffer = dBuffer_[cuda_device_id];

    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;

    // Create sparse matrix A in CSR format
    CUSPARSE_CALL(hipsparseCreateCsr(&matA, m, n, total_nnz, RowPtr.data, ColInd.data, csrVal.data,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    // Create dense matrix B
    int ldb = p;
    CUSPARSE_CALL(hipsparseCreateDnMat(&matB, n, p, ldb, dB, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // Create dense matrix C
    int ldc = p;
    CUSPARSE_CALL(hipsparseCreateDnMat(&matC, m, p, ldc, dC, HIP_R_32F, HIPSPARSE_ORDER_ROW));

    // allocate an external buffer if needed
    float alpha = 1.0f;
    float beta  = 0.0f;
    size_t bufferSize = 0;
    CUSPARSE_CALL(hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize));
    resize(dBuffer, bufferSize);

    // execute SpMM
    CUSPARSE_CALL(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer.data));

    // destroy matrix/vector descriptors
    CUSPARSE_CALL(hipsparseDestroySpMat(matA));
    CUSPARSE_CALL(hipsparseDestroyDnMat(matB));
    CUSPARSE_CALL(hipsparseDestroyDnMat(matC));
#endif

    CUSPARSE_CALL(hipsparseDestroy(handle));
    CUSPARSE_CALL(hipsparseDestroyMatDescr(descrX));
}
